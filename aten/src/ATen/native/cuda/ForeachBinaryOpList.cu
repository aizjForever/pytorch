#include <ATen/Dispatch.h>
#include <ATen/native/ForeachUtils.h>
#include <ATen/native/cuda/ForeachFunctors.cuh>

namespace at { namespace native {

template<template<class> class Op>
std::vector<Tensor> foreach_tensor_list_op(TensorList tensors1, TensorList tensors2, Scalar alpha = 1) {
    std::vector<std::vector<at::Tensor>> tensor_lists; 
    std::vector<at::Tensor> vec_res;
    for (const auto& t: tensors1) {
        vec_res.emplace_back(at::native::empty_like(t));
    }

    tensor_lists.emplace_back(std::move(tensors1.vec()));
    tensor_lists.emplace_back(std::move(tensors2.vec()));
    tensor_lists.emplace_back(std::move(vec_res));

    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kBool, kBFloat16, kHalf, tensors1[0].scalar_type(), "foreach_binary_op_list_cuda", [&]() {
        multi_tensor_apply<3>(tensor_lists, BinaryOpListAlphaFunctor<scalar_t, Op>(), alpha.to<scalar_t>());
    });

    return tensor_lists[2];
}

template<template<class> class Op>
void foreach_tensor_list_op_(TensorList tensors1, TensorList tensors2, Scalar alpha = 1) {
    std::vector<std::vector<at::Tensor>> tensor_lists; 
    tensor_lists.emplace_back(std::move(tensors1.vec()));
    tensor_lists.emplace_back(std::move(tensors2.vec()));

    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kBool, kBFloat16, kHalf, tensors1[0].scalar_type(), "foreach_binary_op_list_cuda_", [&]() {
        multi_tensor_apply<2>(tensor_lists, BinaryOpListAlphaFunctor_<scalar_t, Op>(), alpha.to<scalar_t>());
    });
}

std::vector<Tensor> foreach_tensor_add_list_kernel_cuda(TensorList tensors1, TensorList tensors2, Scalar alpha) {
    verify_list(tensors1, tensors2);
    if (!check_fast_route(tensors1, tensors2)) {
        return at::native::foreach_tensor_add_list_kernel_slow(tensors1, tensors2, alpha);
    }

    return foreach_tensor_list_op<std::plus>(tensors1, tensors2, alpha);
}

void foreach_tensor_add_list_kernel_cuda_(TensorList tensors1, TensorList tensors2, Scalar alpha) {
    verify_list(tensors1, tensors2);

    if (!check_fast_route(tensors1, tensors2)) {
        return at::native::foreach_tensor_add_list_kernel_slow_(tensors1, tensors2, alpha);
    }

    foreach_tensor_list_op_<std::plus>(tensors1, tensors2, alpha);
}

std::vector<Tensor> foreach_tensor_sub_list_kernel_cuda(TensorList tensors1, TensorList tensors2, Scalar alpha) {
    verify_list(tensors1, tensors2);

    if (!check_fast_route(tensors1, tensors2)) {
        return at::native::foreach_tensor_sub_list_kernel_slow(tensors1, tensors2, alpha);
    }

    return foreach_tensor_list_op<std::minus>(tensors1, tensors2, alpha);
}

void foreach_tensor_sub_list_kernel_cuda_(TensorList tensors1, TensorList tensors2, Scalar alpha) {
    verify_list(tensors1, tensors2);

    if (!check_fast_route(tensors1, tensors2)) {
        return at::native::foreach_tensor_sub_list_kernel_slow_(tensors1, tensors2, alpha);
    }

    foreach_tensor_list_op_<std::minus>(tensors1, tensors2, alpha);
}

std::vector<Tensor> foreach_tensor_mul_list_kernel_cuda(TensorList tensors1, TensorList tensors2) {
    verify_list(tensors1, tensors2);

    if (!check_fast_route(tensors1, tensors2)) {
        return at::native::foreach_tensor_mul_list_kernel_slow(tensors1, tensors2);
    }

    return foreach_tensor_list_op<std::multiplies>(tensors1, tensors2);
}

void foreach_tensor_mul_list_kernel_cuda_(TensorList tensors1, TensorList tensors2) {
    verify_list(tensors1, tensors2);

    if (!check_fast_route(tensors1, tensors2)) {
        return at::native::foreach_tensor_mul_list_kernel_slow_(tensors1, tensors2);
    }

    foreach_tensor_list_op_<std::multiplies>(tensors1, tensors2);
}

std::vector<Tensor> foreach_tensor_div_list_kernel_cuda(TensorList tensors1, TensorList tensors2) {
    verify_list(tensors1, tensors2);

    if (!check_fast_route(tensors1, tensors2)) {
        return at::native::foreach_tensor_div_list_kernel_slow(tensors1, tensors2);
    }

    return foreach_tensor_list_op<std::divides>(tensors1, tensors2);
}

void foreach_tensor_div_list_kernel_cuda_(TensorList tensors1, TensorList tensors2) {
    verify_list(tensors1, tensors2);

    if (!check_fast_route(tensors1, tensors2)) {
        return at::native::foreach_tensor_div_list_kernel_slow_(tensors1, tensors2);
    }

    foreach_tensor_list_op_<std::divides>(tensors1, tensors2);
}

}} // namespace at::native
